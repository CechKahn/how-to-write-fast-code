#include "hip/hip_runtime.h"
/*
		Copyright (C) 2011  Abhinav Jauhri (abhinav.jauhri@gmail.com), Carnegie Mellon University - Silicon Valley 

		This program is free software: you can redistribute it and/or modify
		it under the terms of the GNU General Public License as published by
		the Free Software Foundation, either version 3 of the License, or
		(at your option) any later version.

		This program is distributed in the hope that it will be useful,
		but WITHOUT ANY WARRANTY; without even the implied warranty of
		MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
		GNU General Public License for more details.

		You should have received a copy of the GNU General Public License
		along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/


#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "matrix_mul.h"
#include "stdio.h"

#define TILE_WIDTH 2
#define MAX_BLK_DIM 32

#define OPTIMIZE_V1
//#define OPTIMIZE_V2
namespace cuda
{
	__global__ 
	void 
	matrix_mul_kernel(float *sq_matrix_1, float *sq_matrix_2, float *sq_matrix_result, int sq_dimension)
	{
		
		int tx = threadIdx.x;
		int ty = threadIdx.y;
		
		float sum = 0.0f;
		
		for(int k = 0; k < sq_dimension; k++)
		{
			sum += sq_matrix_1[ty*sq_dimension + k] * sq_matrix_2[k*sq_dimension + tx];
		}
		sq_matrix_result[ty*sq_dimension + tx] = sum;
		
	}

	__global__
	void
	matrixMulV1(float *sq_m1, float *sq_m2, float *sq_m3, int sq_dim)
	{
		int row = blockIdx.y * blockDim.y + threadIdx.y;
		int col = blockIdx.x * blockDim.x + threadIdx.x;
		
		float sum = 0.0f;
		if(row >= sq_dim || col >= sq_dim)
			return ;
		int i = 0;
		for(i = 0;i < sq_dim;i++)
		{
			sum+=sq_m1[row * sq_dim + i] * sq_m2[i*sq_dim + col];
		}
		sq_m3[row * sq_dim + col] = sum;
	}

	__global__
	void
	matrixMulV2(float *sq_m1, float *sq_m2, float *sq_m3, int sq_dim)
	{
		int row = blockIdx.y * blockDim.y + threadIdx.y;
		int col = blockIdx.x * blockDim.x + threadIdx.x;
		float sum = 0.0f;
		if(row >= sq_dim || col >= sq_dim)
			return ;
		int i = 0;
		for(i = 0; i < sq_dim;i++)
		{
			sum += sq_m1[row*sq_dim + i] * sq_m2[col*sq_dim + i];
		}
		sq_m3[row * sq_dim + col] = sum;
		return ;
	}

	//this one should try to use tiled matrix multiplication techniques
	//together with shared memory

	__global__
	void
	matrixMulV3(float *sq_m1, float *sq_m2, float *sq_m3, int sq_dim)
	{
		int row = blockIdx.y * blockDim.y + threadIdx.y;
		int col = blockIdx.x * blockDim.x + threadIdx.x;
		
		float sum = 0.0f;
		if(row >= sq_dim || col >= sq_dim)
			return ;
		int i = 0;
		for(i = 0;i < sq_dim;i++)
		{
			sum+=sq_m1[row * sq_dim + i] * sq_m2[i*sq_dim + col];
		}
		sq_m3[row * sq_dim + col] = sum;
	}
	
	void 
	matrix_multiplication(float *sq_matrix_1, float *sq_matrix_2, float *sq_matrix_result, unsigned int sq_dimension)
	{
		int size = sq_dimension * sq_dimension * sizeof(float);
		float *sq_matrix_1_d, *sq_matrix_2_d, *sq_matrix_result_d;
		/***************************************************
	1st Part: Allocation of memory on device memory  
		****************************************************/
		
		/* copy sq_matrix_1 and sq_matrix_2 to device memory */
		hipMalloc((void**) &sq_matrix_1_d, size);
		hipMemcpy(sq_matrix_1_d, sq_matrix_1, size, hipMemcpyHostToDevice);
		hipMalloc((void**) &sq_matrix_2_d, size);
		
		/*allocate sq_matrix_result on host */
		hipMalloc((void**) &sq_matrix_result_d, size);
		
		/***************************************************
	 2nd Part: Inovke kernel 
		****************************************************/
		if(sq_dimension <= 32)
		{
			hipMemcpy(sq_matrix_2_d, sq_matrix_2, size, hipMemcpyHostToDevice);
			dim3 dimBlock(sq_dimension,sq_dimension);
			dim3 dimGrid(1,1);
			matrix_mul_kernel<<<dimGrid,dimBlock>>>(sq_matrix_1_d, sq_matrix_2_d, sq_matrix_result_d, sq_dimension);
		}
		else
		{
#ifdef OPTIMIZE_V1
			//CUDA Version 1, only correctness is ensured here
			hipMemcpy(sq_matrix_2_d, sq_matrix_2, size, hipMemcpyHostToDevice);
			int blockSizeX = MAX_BLK_DIM, blockSizeY = MAX_BLK_DIM;
			dim3 dimBlock(blockSizeX, blockSizeY);
			int gridSizeX = (sq_dimension / blockSizeX) + 1, gridSizeY = (sq_dimension/blockSizeY) + 1;
			dim3 dimGrid(gridSizeX,gridSizeY);
			matrixMulV1<<<dimGrid, dimBlock, sq_dimension * sizeof(float)>>>(sq_matrix_1_d, sq_matrix_2_d, sq_matrix_result_d, sq_dimension);
#endif

#ifdef OPTIMIZE_V2
			dim3 dimBlock(MAX_BLK_DIM,MAX_BLK_DIM);
			int gridSizeX = (sq_dimension / dimBlock.x) + 1, gridSizeY = (sq_dimension/dimBlock.y) + 1;
			dim3 dimGrid(gridSizeX, gridSizeY);
			//transpose the second matrix
			int i = 0, j = 0;
			float *sq_matrix_2t = (float*) malloc(size * sizeof(float));
			float tmp = 0.0f;
			for(i = 0; i < sq_dimension;i++)
			{
				for(j = 0;j < sq_dimension;j++)
				{
					sq_matrix_2t[j * sq_dimension + i] = sq_matrix_2[i * sq_dimension + j];
				}
			}
			hipMemcpy(sq_matrix_2_d, sq_matrix_2t, size, hipMemcpyHostToDevice);
			free(sq_matrix_2t);
			matrixMulV2<<<dimGrid, dimBlock>>>(sq_matrix_1_d, sq_matrix_2_d, sq_matrix_result_d, sq_dimension);
#endif
		}
		/***************************************************
	 3rd Part: Transfer result from device to host 
		****************************************************/
		hipMemcpy(sq_matrix_result, sq_matrix_result_d, size, hipMemcpyDeviceToHost);
		hipFree(sq_matrix_1_d);
		hipFree(sq_matrix_2_d);
		hipFree(sq_matrix_result_d);
	}  
} // namespace cuda
